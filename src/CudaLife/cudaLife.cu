#include "hip/hip_runtime.h"
#include "cudaLife.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include <algorithm>

// Cuenta las 8 vecinas (no incluye la celda central)
__device__ inline ubyte countAliveCells(const ubyte* data,
                                        int x0, int x1, int x2,
                                        int y0, int y1, int y2,
                                        size_t width) {
    return data[y0 * width + x0] + data[y0 * width + x1] + data[y0 * width + x2] +
           data[y1 * width + x0]                         + data[y1 * width + x2] +
           data[y2 * width + x0] + data[y2 * width + x1] + data[y2 * width + x2];
}

__global__ void gameOfLifeKernel(const ubyte* d_data,
                                 ubyte* d_result,
                                 size_t width, size_t height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = width * height;
    if (idx >= total) return;

    int x  = idx % width;
    int y  = idx / width;
    int x0 = (x + width  - 1) % width;
    int x2 = (x + 1) % width;
    int y0 = (y + height - 1) % height;
    int y2 = (y + 1) % height;

    ubyte alive = countAliveCells(d_data, x0, x, x2, y0, y, y2, width);
    d_result[idx] = (alive == 3 || (alive == 2 && d_data[idx])) ? 1 : 0;
}

// Kernel 2D para el Juego de la Vida, cada thread procesa una celda (x,y)
__global__ void gameOfLifeKernel2D(const ubyte* d_data, ubyte* d_result, size_t width, size_t height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    // Coordenadas con borde periódico
    int x0 = (x + width  - 1) % width;
    int x2 = (x + 1) % width;
    int y0 = (y + height - 1) % height;
    int y2 = (y + 1) % height;

    // Cuenta las células vivas vecinas y aplica reglas del juego
    ubyte alive = countAliveCells(d_data, x0, x, x2, y0, y, y2, width);
    d_result[y * width + x] = (alive == 3 || (alive == 2 && d_data[y * width + x])) ? 1 : 0;
}



void computeIterationGPU(ubyte*& d_data,
                         ubyte*& d_result,
                         size_t width, size_t height, int threadsPerBlock) {
    size_t total = width * height;
    int blocks = (total + threadsPerBlock - 1) / threadsPerBlock;

    gameOfLifeKernel<<<blocks, threadsPerBlock>>>(d_data, d_result, width, height);
    hipDeviceSynchronize();

    // swap para la siguiente iteración
    std::swap(d_data, d_result);
}

void computeIterationGPU2D(ubyte*& d_data, ubyte*& d_result, size_t width, size_t height) {
    dim3 threads(32, 32);
    dim3 blocks((width + threads.x - 1) / threads.x,
                (height + threads.y - 1) / threads.y);

    gameOfLifeKernel2D<<<blocks, threads>>>(d_data, d_result, width, height);
    hipDeviceSynchronize();
    std::swap(d_data, d_result);
}


void initWorld(ubyte*& d_data, ubyte*& d_result,
               size_t width, size_t height) {
    size_t bytes = width * height * sizeof(ubyte);
    hipMalloc(&d_data,   bytes);
    hipMalloc(&d_result, bytes);
}

void cleanupWorld(ubyte* d_data, ubyte* d_result) {
    hipFree(d_data);
    hipFree(d_result);
}